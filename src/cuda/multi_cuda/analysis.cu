#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <array>
#include "../cuda_common.cuh"

#include <timing.h>
#include "analysis.h"

#define THREAD_COUNT 1024
#define EXTRA_WORKLISTS 50
#define COLLISIONS_BEFORE_SWITCH (1)

using namespace multi_cuda;

__device__ inline Node& get_node(Node* nodes, int index, int node_size){
    int8_t* byte_ptr = (int8_t*)nodes;
    int byte_offset = index * node_size;
    
    return *(Node*)(byte_ptr + byte_offset);
}

__device__ void add_sucessors_to_worklist(int* successors, int work_columns[][THREAD_COUNT], int work_column_count, int current_work_column, int* worklists_pending){
    int initial_work_column = current_work_column;
    for(int i = 0; i < 5; i++){
        current_work_column = initial_work_column;
        int amount_of_new_worklists = 1;
        int succ_index = successors[i];
        if (succ_index == -1)
            return;
        unsigned long hash = succ_index * 120811; 
        int collision_count = 0;
        int* work_column = work_columns[current_work_column];
        while(atomicCAS(&work_column[hash % THREAD_COUNT], -1, succ_index) != -1){
            if(work_column[hash % THREAD_COUNT] == succ_index){
                break;
            }
            
            if(++collision_count >= COLLISIONS_BEFORE_SWITCH){
                current_work_column = (current_work_column + 1) % work_column_count;
                work_column = work_columns[current_work_column];
                amount_of_new_worklists++;
                collision_count = 0;
            }else{
                hash++;
            }
        }
        
        atomicMax(worklists_pending, amount_of_new_worklists);
    }
}

__device__ BitVector multi_cuda_join(int predecessors[], Node *nodes, int node_size, int source_index){
        BitVector joined_data = 0;
        int pred_index = 0;
        while (predecessors[pred_index] != -1){
            joined_data |= get_node(nodes, predecessors[pred_index], node_size).data[source_index];
            ++pred_index;
        }
        return joined_data;
}

__global__ void analyze(Node* nodes, int work_columns[][THREAD_COUNT], int work_column_count, Transfer transfers[], int node_count, int* worklists_pending, int current_work_column, int source_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    int* work_column = work_columns[current_work_column];
    int node_size = sizeof(Node) + sizeof(BitVector) * source_count;

    if(node_index < THREAD_COUNT && work_column[node_index] != -1){
        Node& current_node = get_node(nodes, work_column[node_index], node_size);

        bool add_successors = false;

        for(int source = 0; source < source_count; ++source){
            BitVector last = current_node.data[source];
            BitVector current = last;


            BitVector joined_data = multi_cuda_join(current_node.predecessor_index, nodes, node_size, source);
            current |= joined_data & current_node.join_mask;
            
            joined_data |= current;
            transfer_function(current_node.first_transfer_index, transfers, joined_data, current);

            if(last != current){
                current_node.data[source] = current;
                add_successors = true;
            }
        }

        if(add_successors){
            add_sucessors_to_worklist(current_node.successor_index, work_columns, work_column_count, (current_work_column+1) % work_column_count, worklists_pending);
        }

        work_column[node_index] = -1;   
    }
}

void multi_cuda::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count, std::set<int>& taint_sources, int source_count) {
    Node* dev_nodes = nullptr;
    Transfer* dev_transfers = nullptr;
    int** dev_worklists = nullptr;
    int* dev_worklists_pending = nullptr;


    int worklists_pending = ((node_count + THREAD_COUNT - 1)/THREAD_COUNT);
    int threadsPerBlock = 128;
    int block_count = THREAD_COUNT/threadsPerBlock;    
    int work_column_count = worklists_pending + EXTRA_WORKLISTS;

    int node_size = sizeof(BitVector) * source_count + sizeof(Node);

    std::vector<std::array<int, THREAD_COUNT>> worklists{};

    std::set<int>::iterator it = taint_sources.begin();
    for(int i = 0; i < work_column_count; i++){
        worklists.emplace_back(); 
        for(int j = 0; j < THREAD_COUNT; j++){
            if(it != taint_sources.end()) {
                worklists[i][j] = *it;
                it++;
            }else{
                worklists[i][j] = -1;
            }
        }
    }

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return;
    }

    // Allocate work columns
    dev_worklists = cuda_allocate_memory<int*>(sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    hipMemset(dev_worklists, -1, sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    cuda_copy_to_device(dev_worklists, &worklists[0], sizeof(int) * THREAD_COUNT * work_column_count);

    // Allocate GPU buffers for three vectors (two input, one output)  
    dev_nodes = cuda_allocate_memory<Node>(node_size * node_count + 1);
    cuda_copy_to_device(dev_nodes, nodes, node_size * node_count);

    dev_worklists_pending = (int*) (dev_nodes + node_count);

    // Allocate transfer function
    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_transfers, transfers, sizeof(Transfer)*transfer_count);
  
    Stopwatch lfp_watch;
    int current_worklist = 0;
    while(worklists_pending > 0){
        --worklists_pending;
        cuda_copy_to_device(dev_worklists_pending, &worklists_pending, sizeof(int));

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, (int(*)[THREAD_COUNT])dev_worklists, work_column_count+1, dev_transfers, node_count, dev_worklists_pending, current_worklist, source_count);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }
        
        cuda_copy_to_host((void*)&worklists_pending, dev_worklists_pending, sizeof(int));
        current_worklist = (current_worklist+1) % work_column_count;
    }
    lfp_watch.print_time<Microseconds>("LFP time: ");


    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, node_size*node_count);

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
    cuda_free(dev_worklists);
}