#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <array>
#include "../cuda_common.cuh"

#include <timing.h>
#include "analysis.h"

#define THREAD_COUNT 1024
#define COLLISIONS_BEFORE_SWITCH (1)

using namespace multi_cuda;

__device__ inline Node& get_node(Node* nodes, int index, int node_size){
    int8_t* byte_ptr = (int8_t*)nodes;
    int byte_offset = index * node_size;
    
    return *(Node*)(byte_ptr + byte_offset);
}

__device__ void add_sucessors_to_worklist(int* successors, int work_columns[][THREAD_COUNT], int work_column_count, int current_work_column){
    int initial_work_column = current_work_column;
    for(int i = 0; i < 5; i++){
        int succ_index = successors[i];
        if (succ_index == -1)
            return;
        intptr_t hash = succ_index * 120811; 
        int collision_count = 0;
        int* work_column = work_columns[current_work_column];
        while(work_column[hash % THREAD_COUNT] != -1){
            if(work_column[hash % THREAD_COUNT] == succ_index){
                break;
            }
            

            if(++collision_count >= COLLISIONS_BEFORE_SWITCH){
                current_work_column = (current_work_column + 1) % work_column_count;
                work_column = work_columns[current_work_column];
            }else{
                hash++;
            }
        }
        work_column[hash % THREAD_COUNT] = succ_index;
    }
}

__device__ BitVector multi_cuda_join(int predecessors[], Node *nodes, int node_size, int source_index){
        BitVector joined_data = 0;
        int pred_index = 0;
        while (predecessors[pred_index] != -1){
            joined_data |= get_node(nodes, predecessors[pred_index], node_size).data[source_index];
            ++pred_index;
        }
        return joined_data;
}

__global__ void analyze(Node* nodes, int work_columns[][THREAD_COUNT], int work_column_count, Transfer transfers[], int node_count, bool* work_to_do, int i, int source_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    int* work_column = work_columns[i];
    int node_size = sizeof(Node) + sizeof(BitVector) * source_count;

    if(node_index < THREAD_COUNT && work_column[node_index] != -1){
        int index = node_size * work_column[node_index];
        Node& current_node = get_node(nodes, work_column[node_index], node_size);

        bool add_successors = false;

        for(int source = 0; source < source_count; ++source){
            BitVector last = current_node.data[source];
            BitVector current = last;


            BitVector joined_data = multi_cuda_join(current_node.predecessor_index, nodes, node_size, source);
            current |= joined_data & current_node.join_mask;
            
            joined_data |= current;
            transfer_function(current_node.first_transfer_index, transfers, joined_data, current);

            if(last != current){
                current_node.data[source] = current;
                add_successors = true;
            }
        }

        if(add_successors){
            add_sucessors_to_worklist(current_node.successor_index, work_columns, work_column_count, (i+1) % work_column_count);
            *work_to_do = true;
        }

        work_column[node_index] = -1;   
    }
}

void multi_cuda::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count, std::set<int>& taint_sources, int source_count) {
    Node* dev_nodes = nullptr;
    bool* dev_work_to_do = nullptr;
    Transfer* dev_transfers = nullptr;
    int** dev_worklists = nullptr;

    bool work_to_do = true;
    int threadsPerBlock = 128;
    int block_count = THREAD_COUNT/threadsPerBlock;    
    int work_column_count = ((node_count + THREAD_COUNT - 1)/THREAD_COUNT) + 50;

    int node_size = sizeof(BitVector) * source_count + sizeof(Node);

    std::vector<std::array<int, THREAD_COUNT>> worklists{};

    std::set<int>::iterator it = taint_sources.begin();
    for(int i = 0; i < work_column_count; i++){
        worklists.emplace_back(); 
        for(int j = 0; j < THREAD_COUNT; j++){
            if(it != taint_sources.end()) {
                worklists[i][j] = *it;
                it++;
            }else{
                worklists[i][j] = -1;
            }
        }
    }

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return;
    }

    // Allocate work columns
    dev_worklists = cuda_allocate_memory<int*>(sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    hipMemset(dev_worklists, -1, sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    cuda_copy_to_device(dev_worklists, &worklists[0], sizeof(int) * THREAD_COUNT * work_column_count);

    // Allocate GPU buffers for three vectors (two input, one output)  
    dev_nodes = cuda_allocate_memory<Node>(node_size * node_count + 1);
    cuda_copy_to_device(dev_nodes, nodes, node_size * node_count);

    dev_work_to_do = (bool*) (dev_nodes + node_count);

    // Allocate transfer function
    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_transfers, transfers, sizeof(Transfer)*transfer_count);
  
    Stopwatch lfp_watch;
    int i = 0;
    while(work_to_do){
        work_to_do = false;
        cuda_copy_to_device(dev_work_to_do, &work_to_do, 1);

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, (int(*)[THREAD_COUNT])dev_worklists, work_column_count+1, dev_transfers, node_count, dev_work_to_do, i, source_count);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }
        
        cuda_copy_to_host((void*)&work_to_do, dev_work_to_do, sizeof(bool));
        i = (i+1) % (work_column_count+1);
    }
    lfp_watch.print_time<Microseconds>("LFP time: ");


    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, node_size*node_count);

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
    cuda_free(dev_worklists);
}