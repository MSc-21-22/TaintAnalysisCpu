#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <array>
#include "../cuda_common.cuh"

#include <timing.h>
#include "analysis.h"

#define THREAD_COUNT 1024
#define COLLISIONS_BEFORE_SWITCH (1) 

using namespace cuda_worklist;

__device__ void add_sucessors_to_worklist(int* successors, int work_columns[][THREAD_COUNT], int work_column_count, int current_work_column, Node* nodes){
    int initial_work_column = current_work_column;
    for(int i = 0; i < 5; i++){
        int succ_index = successors[i];
        if (succ_index == -1)
            return;
        intptr_t hash = reinterpret_cast<intptr_t>(&nodes[succ_index]);
        int collision_count = 0;
        int* work_column = work_columns[current_work_column];

        int old;
        do{
            while(work_column[hash % THREAD_COUNT] != -1){
                if(work_column[hash % THREAD_COUNT] == succ_index){
                    break;
                }
                

                if(++collision_count >= COLLISIONS_BEFORE_SWITCH){
                    current_work_column = (current_work_column + 1) % work_column_count;
                    work_column = work_columns[current_work_column];
                }else{
                    hash++;
                }
            }
            // work_column[hash % THREAD_COUNT] = succ_index;

            old = atomicCAS(&work_column[hash % THREAD_COUNT], -1, succ_index);
        }while (old == -1);
    }
}

__global__ void analyze(Node nodes[], int work_columns[][THREAD_COUNT], int work_column_count, Transfer transfers[], int node_count, bool* work_to_do, int i){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    int* work_column = work_columns[i];

    if(node_index < THREAD_COUNT && work_column[node_index] != -1){
        Node& current_node = nodes[work_column[node_index]];
        
        BitVector last = current_node.data;
        BitVector current = current_node.data;

        BitVector joined_data = join(current_node.predecessor_index, nodes);
        current |= joined_data & current_node.join_mask;

        transfer_function(current_node.first_transfer_index, transfers, joined_data, current);

        if(last != current){
            current_node.data = current;
            add_sucessors_to_worklist(current_node.successor_index, work_columns, work_column_count, (i+1) % work_column_count, nodes);
            *work_to_do = true;
        }
        work_column[node_index] = -1;   
    }
}

void cuda_worklist::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count, std::set<int>& taint_sources) {
    Node* dev_nodes = nullptr;
    bool* dev_work_to_do = nullptr;
    Transfer* dev_transfers = nullptr;
    int** dev_worklists = nullptr;

    bool work_to_do = true;
    int threadsPerBlock = 128;
    int block_count = THREAD_COUNT/threadsPerBlock;    
    int work_column_count = ((node_count + THREAD_COUNT - 1)/THREAD_COUNT) + 50;

    std::vector<std::array<int, THREAD_COUNT>> worklists{};

    std::set<int>::iterator it = taint_sources.begin();
    for(int i = 0; i < work_column_count; i++){
        worklists.emplace_back(); 
        for(int j = 0; j < THREAD_COUNT; j++){
            if(it != taint_sources.end()) {
                worklists[i][j] = *it;
                it++;
            }else{
                worklists[i][j] = -1;
            }
        }
    }

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return;
    }

    // Allocate work columns
    dev_worklists = cuda_allocate_memory<int*>(sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    hipMemset(dev_worklists, -1, sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    cuda_copy_to_device(dev_worklists, &worklists[0], sizeof(int) * THREAD_COUNT * work_column_count);

    // Allocate GPU buffers for three vectors (two input, one output)  
    dev_nodes = cuda_allocate_memory<Node>(sizeof(Node)*node_count + 1);
    cuda_copy_to_device(dev_nodes, nodes, sizeof(Node)*node_count);

    dev_work_to_do = (bool*) (dev_nodes + node_count);

    // Allocate transfer function
    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_transfers, transfers, sizeof(Transfer)*transfer_count);
  
    Stopwatch lfp_watch;
    int i = 0;
    while(work_to_do){
        work_to_do = false;
        cuda_copy_to_device(dev_work_to_do, &work_to_do, 1);

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, (int(*)[THREAD_COUNT])dev_worklists, work_column_count+1, dev_transfers, node_count, dev_work_to_do, i);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }
        
        cuda_copy_to_host((void*)&work_to_do, dev_work_to_do, sizeof(bool));
        i = (i+1) % (work_column_count+1);
    }
    lfp_watch.print_time<Microseconds>("LFP time: ");


    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, sizeof(Node)*node_count);

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
    cuda_free(dev_worklists);
}