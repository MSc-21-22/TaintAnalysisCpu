#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <array>
#include "../cuda_common.cuh"

#include <timing.h>
#include "analysis.h"

#define THREAD_COUNT 1024
#define EXTRA_WORKLISTS 50
#define COLLISIONS_BEFORE_SWITCH (1) 

using namespace cuda_worklist;

__device__ void add_sucessors_to_worklist(int* successors, int work_columns[][THREAD_COUNT], int work_column_count, int initial_work_column, Node* nodes, int* worklists_pending){
    int current_work_column;
    for(int i = 0; i < 5; i++){
        int amount_of_new_worklists = 1;
        current_work_column = initial_work_column;
        int succ_index = successors[i];
        if (succ_index == -1)
            return;
        unsigned long hash = succ_index*120811;
        int collision_count = 0;
        int* work_column = work_columns[current_work_column];

        while(atomicCAS(&work_column[hash % THREAD_COUNT], -1, succ_index) != -1){
            if(work_column[hash % THREAD_COUNT] == succ_index){
                break;
            }
            
            if(++collision_count >= COLLISIONS_BEFORE_SWITCH){
                current_work_column = (current_work_column + 1) % work_column_count;
                work_column = work_columns[current_work_column];
                amount_of_new_worklists++;
                collision_count = 0;
            }else{
                hash++;
            }
        }
        
        atomicMax(worklists_pending, amount_of_new_worklists);
    }
}

__global__ void analyze(Node nodes[], int work_columns[][THREAD_COUNT], int work_column_count, Transfer transfers[], int node_count, int* worklists_pending, int current_work_column){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    int* work_column = work_columns[current_work_column];

    if(node_index < THREAD_COUNT && work_column[node_index] != -1){
        Node& current_node = nodes[work_column[node_index]];
        
        BitVector last = current_node.data;
        BitVector current = current_node.data;

        BitVector joined_data = join(current_node.predecessor_index, nodes);
        current |= joined_data & current_node.join_mask;

        transfer_function(current_node.first_transfer_index, transfers, joined_data, current);

        if(last != current){
            current_node.data = current;
            int next_work_column = (current_work_column+1) % work_column_count;
            add_sucessors_to_worklist(current_node.successor_index, work_columns, work_column_count, next_work_column, nodes, worklists_pending);
        }
        
        work_column[node_index] = -1;   
    }
}

void cuda_worklist::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count, std::set<int>& taint_sources) {
    Node* dev_nodes = nullptr;
    int* dev_worklists_pending = nullptr;
    Transfer* dev_transfers = nullptr;
    int** dev_worklists = nullptr;

    int worklists_pending = ((node_count + THREAD_COUNT - 1)/THREAD_COUNT);
    int threadsPerBlock = 128;
    int block_count = THREAD_COUNT/threadsPerBlock;    
    int work_column_count = worklists_pending + EXTRA_WORKLISTS;

    std::vector<std::array<int, THREAD_COUNT>> worklists{};

    std::set<int>::iterator it = taint_sources.begin();
    for(int i = 0; i < work_column_count; i++){
        worklists.emplace_back(); 
        for(int j = 0; j < THREAD_COUNT; j++){
            if(it != taint_sources.end()) {
                worklists[i][j] = *it;
                it++;
            }else{
                worklists[i][j] = -1;
            }
        }
    }

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return;
    }

    // Allocate work columns
    dev_worklists = cuda_allocate_memory<int*>(sizeof(int) * THREAD_COUNT * work_column_count);
    hipMemset(dev_worklists, -1, sizeof(int) * THREAD_COUNT * work_column_count);
    cuda_copy_to_device(dev_worklists, &worklists[0], sizeof(int) * THREAD_COUNT * work_column_count);

    // Allocate GPU buffers for three vectors (two input, one output)  
    dev_nodes = cuda_allocate_memory<Node>(sizeof(Node)*node_count + sizeof(int));
    cuda_copy_to_device(dev_nodes, nodes, sizeof(Node)*node_count);

    dev_worklists_pending = (int*) (dev_nodes + node_count);

    // Allocate transfer function
    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_transfers, transfers, sizeof(Transfer)*transfer_count);
  
    Stopwatch lfp_watch;
    int current_worklist = 0;
    while(worklists_pending > 0){
        --worklists_pending;
        cuda_copy_to_device(dev_worklists_pending, &worklists_pending, sizeof(int));

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, (int(*)[THREAD_COUNT])dev_worklists, work_column_count+1, dev_transfers, node_count, dev_worklists_pending, current_worklist);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }
        
        cuda_copy_to_host((void*)&worklists_pending, dev_worklists_pending, sizeof(int));
        current_worklist = (current_worklist+1) % work_column_count;
    }
    lfp_watch.print_time<Microseconds>("LFP time: ");


    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, sizeof(Node)*node_count);

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
    cuda_free(dev_worklists);
}