
#include <hip/hip_runtime.h>
#include <stdio.h>

void cuda_allocate_memory(void **devPtr, size_t size){
    auto cudaStatus = hipMalloc(devPtr, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}

void cuda_copy_to_device(void *dst, const void *src, size_t size){
    auto cudaStatus = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to gpu failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}

void cuda_copy_to_host(void *dst, const void *src, size_t size){
    auto cudaStatus = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}

void cuda_free(void* devPtr){
    auto cudaStatus = hipFree(devPtr);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}