#include <stdio.h>
#include <cuda/cuda_common.cuh>
#include <cuda/common.h>

void cuda_copy_to_device(void *dst, const void *src, size_t size){
    auto cudaStatus = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to gpu failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}

void cuda_copy_to_host(void *dst, const void *src, size_t size){
    auto cudaStatus = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}

void cuda_free(void* devPtr){
    auto cudaStatus = hipFree(devPtr);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}

void init_gpu(){
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return;
    }

    int* x = cuda_allocate_memory<int>(4);
    cuda_free(x);
}


std::future<void> init_gpu_async(){
    return std::async(std::launch::async, []{init_gpu();});
}

void cuda_allocate_memory(void** devPtr, size_t size){
    auto cudaStatus = hipMalloc(devPtr, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }
}