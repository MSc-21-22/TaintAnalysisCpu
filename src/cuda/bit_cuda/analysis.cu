#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "../cuda_common.cuh"

#include "analysis.h"

using namespace bit_cuda;

__global__ void analyze(Node nodes[], Transfer transfers[], bool* has_changed, int node_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(node_index == 0)
        *has_changed = true;

    if(node_index < node_count){
        nodes[node_index].data = 1; // Set taint constant to true
        
        bool is_changed = true;
        BitVector last_joined = 0;
        BitVector current = nodes[node_index].data;

        while(*has_changed){
            if(node_index == 0)
                *has_changed = false;
            BitVector joined_data = 1;
            //Join
            {
                int pred_index = 0;
                while (nodes[node_index].predecessor_index[pred_index] != -1){
                    joined_data |= nodes[nodes[node_index].predecessor_index[pred_index]].data;
                    ++pred_index;
                }

                is_changed |= last_joined != joined_data;
                last_joined = joined_data;
                current |= joined_data & nodes[node_index].join_mask;
            }

            //Transfer
            if(is_changed){
                Transfer* transfer;
                int transfer_index = nodes[node_index].first_transfer_index;

                while(transfer_index != -1){
                    transfer = &transfers[transfer_index];
                    int var_index = 0;
                    int next_var = transfer->rhs[var_index];
                    while(next_var != -1){

                        if((joined_data & (1 << next_var)) != 0){
                            current |= (1 << transfer->x);
                            break;
                        }
                        ++var_index;
                        next_var = transfer->rhs[var_index];
                    }
                    transfer_index = transfer->next_transfer_index;
                }
                printf("[%d] [%d] (%d)\n", current, last_joined, node_index);
                nodes[node_index].data = current;
                *has_changed = true;
                is_changed = false;
                // __syncthreads();
            }
            __syncthreads();
            __threadfence();
        }
    }

}

void bit_cuda::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count) {
    Node* dev_nodes = nullptr;
    bool* dev_has_changed = nullptr;
    Transfer* dev_extra_transfers = nullptr;

    int block_count = node_count/128 + 1;
    dim3 threadsPerBlock(128);
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        exit(1);
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    cuda_allocate_memory((void**)&dev_nodes, sizeof(Node)*node_count + 1);
    cuda_copy_to_device(dev_nodes, nodes, sizeof(Node)*node_count);

    cuda_allocate_memory((void**)&dev_extra_transfers, sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_extra_transfers, transfers, sizeof(Transfer)*transfer_count);
    
    dev_has_changed = (bool*) (dev_nodes + (sizeof(Node)*node_count));

    // Launch a kernel on the GPU with one thread for each element.
    analyze<<<block_count, threadsPerBlock>>>(dev_nodes, dev_extra_transfers, dev_has_changed, node_count);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        exit(1);
    }

    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, sizeof(Node)*node_count);
}