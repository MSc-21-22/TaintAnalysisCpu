#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "../cuda_common.cuh"

#include "analysis.h"

using namespace bit_cuda;

__global__ void analyze(Node nodes[], Transfer transfers[], bool* has_changed, int node_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(node_index < node_count){
        Node& current_node = nodes[node_index];
        
        BitVector current = current_node.data;
        BitVector last = current_node.data;
        
        BitVector joined_data = join(current_node.predecessor_index, nodes);
        current |= joined_data & current_node.join_mask;

        transfer_function(current_node.first_transfer_index, transfers, joined_data, current);

        if(last != current){
            current_node.data = current;
            *has_changed = true;
        }
    }
}

void bit_cuda::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count) {
    Node* dev_nodes = nullptr;
    bool* dev_has_changed = nullptr;
    Transfer* dev_transfers = nullptr;

    int block_count = node_count/128 + 1;
    dim3 threadsPerBlock(128);
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        exit(1);
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    dev_nodes = cuda_allocate_memory<Node>(sizeof(Node)*node_count + 1);
    cuda_copy_to_device(dev_nodes, nodes, sizeof(Node)*node_count);

    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_transfers, transfers, sizeof(Transfer)*transfer_count);
    
    dev_has_changed = (bool*) (dev_nodes + node_count);

    bool has_changed = true;
    while(has_changed){
        has_changed = false;
        cuda_copy_to_device(dev_has_changed, &has_changed, sizeof(bool));

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, dev_transfers, dev_has_changed, node_count);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(1);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }

        cuda_copy_to_host((void*)&has_changed, dev_has_changed, sizeof(bool));
    }


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        exit(1);
    }

    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, sizeof(Node)*node_count);

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
}