#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "../cuda_common.cuh"

#include "analysis.h"

using namespace bit_cuda;

__device__ BitVector join(int predecessors[], Node nodes[]){
        BitVector joined_data = 1;
        int pred_index = 0;
        while (predecessors[pred_index] != -1){
            joined_data |= nodes[predecessors[pred_index]].data;
            ++pred_index;
        }
        return joined_data;
}

__device__ void transfer_function_bit_cuda(int first_transfer_index, Transfer transfers[], BitVector& joined_data, BitVector& current){
    Transfer* transfer;
    int transfer_index = first_transfer_index;

    while(transfer_index != -1){
        transfer = &transfers[transfer_index];
        int var_index = 0;
        int next_var = transfer->rhs[var_index];
        while(next_var != -1){

            if((joined_data & (1 << next_var)) != 0){
                current |= (1 << transfer->x);
                break;
            }
            ++var_index;
            next_var = transfer->rhs[var_index];
        }
        transfer_index = transfer->next_transfer_index;
    }
}

__global__ void analyze(Node nodes[], Transfer transfers[], bool* has_changed, int node_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(node_index < node_count){
        Node& current_node = nodes[node_index];
        
        BitVector current = current_node.data;
        BitVector last = current_node.data;
        
        BitVector joined_data = join(current_node.predecessor_index, nodes);
        current |= joined_data & current_node.join_mask;

        transfer_function_bit_cuda(current_node.first_transfer_index, transfers, joined_data, current);

        if(last != current){
            current_node.data = current;
            *has_changed = true;
        }
    }
}

void bit_cuda::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count) {
    Node* dev_nodes = nullptr;
    bool* dev_has_changed = nullptr;
    Transfer* dev_transfers = nullptr;

    int block_count = node_count/128 + 1;
    dim3 threadsPerBlock(128);
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        exit(1);
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    dev_nodes = cuda_allocate_memory<Node>(sizeof(Node)*node_count + 1);
    cuda_copy_to_device(dev_nodes, nodes, sizeof(Node)*node_count);

    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfer_count);
    cuda_copy_to_device(dev_transfers, transfers, sizeof(Transfer)*transfer_count);
    
    dev_has_changed = (bool*) (dev_nodes + node_count);

    bool has_changed = true;
    while(has_changed){
        has_changed = false;
        cuda_copy_to_device(dev_has_changed, &has_changed, sizeof(bool));

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, dev_transfers, dev_has_changed, node_count);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(1);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }

        cuda_copy_to_host((void*)&has_changed, dev_has_changed, sizeof(bool));
        std::cout << "has changed: " << has_changed << std::endl;
    }


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        exit(1);
    }

    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(nodes, dev_nodes, sizeof(Node)*node_count);

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
}