#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "../cuda_common.cuh"
#include "timing.h"

#include "analysis.h"

using namespace bit_cuda;
using namespace taint;

__global__ void analyze(Node nodes[], BitVector data[], Transfer transfers[], bool* has_changed, int node_count) {
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(node_index < node_count){
        Node& current_node = nodes[node_index];
        
        BitVector current = data[node_index];
        BitVector last = data[node_index];
        
        BitVector joined_data = join(current_node.predecessor_index, nodes, data);
        current.bitfield |= joined_data.bitfield & current_node.join_mask.bitfield;

        transfer_function(current_node.first_transfer_index, transfers, joined_data, current);

        if(last.bitfield != current.bitfield){
            data[node_index] = current;
            *has_changed = true;
        }
    }
}

std::vector<BitVector> bit_cuda::execute_analysis(DynamicArray<taint::Node>& nodes, std::vector<Transfer>& transfers) {
    Node* dev_nodes = nullptr;
    bool* dev_has_changed = nullptr;
    Transfer* dev_transfers = nullptr;
    BitVector* dev_data = nullptr;

    int block_count = nodes.size() / 128 + 1;
    dim3 threadsPerBlock(128);
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        exit(1);
    }

    Stopwatch a_node_clock{};
    // Allocate GPU buffers for three vectors (two input, one output)
    dev_nodes = cuda_allocate_memory<Node>(nodes.full_size() + 1);
    a_node_clock.print_time<Microseconds>("Allocate nodes: ");
    Stopwatch c_node_clock{};
    cuda_copy_to_device(dev_nodes, nodes.get_ptr(), nodes.full_size());
    c_node_clock.print_time<Microseconds>("Copy nodes: ");

    Stopwatch a_transfers_clock{};
    dev_transfers = cuda_allocate_memory<Transfer>(sizeof(Transfer)*transfers.size());
    a_transfers_clock.print_time<Microseconds>("Allocate transfers: ");
    Stopwatch c_transfers_clock{};
    cuda_copy_to_device(dev_transfers, transfers.data(), sizeof(Transfer) * transfers.size());
    c_transfers_clock.print_time<Microseconds>("Copy transfers: ");

    Stopwatch a_data_clock{};
    std::vector<BitVector> data(nodes.size(), 1);
    dev_data = cuda_allocate_memory<BitVector>(sizeof(BitVector) * nodes.size());
    a_data_clock.print_time<Microseconds>("Allocate data: ");
    Stopwatch c_data_clock{};
    cuda_copy_to_device(dev_data, data.data(), data.size() * sizeof(BitVector));
    c_data_clock.print_time<Microseconds>("Copy data: ");

    
    dev_has_changed = (bool*) (dev_nodes + nodes.size());

    Stopwatch lfp_clock{};
    bool has_changed = true;
    while(has_changed){
        has_changed = false;
        cuda_copy_to_device(dev_has_changed, &has_changed, sizeof(bool));

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, dev_data, dev_transfers, dev_has_changed, nodes.size());
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(1);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }

        cuda_copy_to_host((void*)&has_changed, dev_has_changed, sizeof(bool));
    }
    lfp_clock.print_time<Microseconds>("Least fixed point: ");


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        exit(1);
    }

    Stopwatch output_clock{};
    // Copy output vector from GPU buffer to host memory.
    cuda_copy_to_host(data.data(), dev_data, sizeof(BitVector) * nodes.size());
    output_clock.print_time<Microseconds>("Copying result to host: ");

    cuda_free(dev_nodes);
    cuda_free(dev_transfers);
    cuda_free(dev_data);

    return data;
}