#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

#include "analysis.h"

using namespace bit_cuda;

__global__ void analyze(Node nodes[], bool* has_changed, int node_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(node_index < node_count){
        nodes[node_index].data.data = 1; // Set taint constant to true

        while(*has_changed){
            *has_changed = false;
            bool is_changed = false;
            long int new_data = 0;
            //Join
            {
                long int old_data = nodes[node_index].data.data;
                new_data = old_data;
                int pred_index = 0;
                while (nodes[node_index].predecessor_index[pred_index] != -1){
                    new_data |= nodes[nodes[node_index].predecessor_index[pred_index]].data.data;
                    ++pred_index;
                }

                is_changed = old_data == new_data;
            }

            //Transfer
            if(is_changed){
                int var_index = 0;
                int next_var = nodes[node_index].transfer.rhs[var_index];
                while(next_var != -1){
                    if(nodes[node_index].data.data & (1 << next_var) > 0){
                        nodes[node_index].data.data |= nodes[node_index].transfer.x;
                        break;
                    }
                    ++var_index;
                    next_var = nodes[node_index].transfer.rhs[var_index];
                }

                *has_changed = true;
            }
        }
    }

}

void bit_cuda::execute_analysis(Node* nodes, int node_count) {
    Node* dev_nodes = nullptr;
    bool* dev_has_changed = nullptr;

    int block_count = node_count/128 + 1;
    dim3 threadsPerBlock(128);

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nodes, sizeof(Node)*node_count + 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    dev_has_changed = (bool*) (dev_nodes + (sizeof(Node)*node_count));

    cudaStatus = hipMemcpy(dev_nodes, nodes, sizeof(Node)*node_count, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy false to gpu failed");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    analyze<<<block_count, threadsPerBlock>>>(dev_nodes, dev_has_changed, node_count);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&nodes, dev_nodes, sizeof(Node)*node_count, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed with message: %d", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_nodes);
}