#include "hip/hip_runtime.h"
#include "analysis.h"

__global__ void analyze(Node nodes[], bool* has_changed){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    nodes[node_index].data.data = 1; // Set taint constant to true

    while(*has_changed){
        *has_changed = false;
        bool is_changed = false;
        long int new_data = 0;
        //Join
        {
            long int old_data = nodes[node_index].data.data;
            new_data = old_data;
            int pred_index = 0;
            while (nodes[node_index].predecessor_index[pred_index] != -1){
                new_data |= nodes[nodes[node_index].predecessor_index[pred_index]].data.data;
                ++pred_index;
            }

            is_changed = old_data == new_data;
        }

        //Transfer
        if(is_changed){
            int var_index = 0;
            int next_var = nodes[node_index].transfer.rhs[var_index];
            while(next_var != -1){
                if(nodes[node_index].data.data & (1 << next_var) > 0){
                    nodes[node_index].data.data |= nodes[node_index].transfer.x;
                    break;
                }
                ++var_index;
                next_var = nodes[node_index].transfer.rhs[var_index];
            }

            *has_changed = true;
        }
    }
}