#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

#include "analysis.h"

using namespace bit_cuda;

__global__ void analyze(Node nodes[], Transfer transfers[], bool* has_changed, int node_count){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(node_index == 0)
        *has_changed = true;

    if(node_index < node_count){
        nodes[node_index].data.data = 1; // Set taint constant to true
        bool is_changed = true;

        while(*has_changed){
            if(node_index == 0)
                *has_changed = false;
            long int new_data = 0;
            //Join
            {
                long int old_data = nodes[node_index].data.data;
                new_data = old_data;
                int pred_index = 0;
                while (nodes[node_index].predecessor_index[pred_index] != -1){
                    __syncthreads();
                    new_data |= nodes[nodes[node_index].predecessor_index[pred_index]].data.data;
                    ++pred_index;
                }

                is_changed |= old_data != new_data;
            }

            //Transfer
            if(is_changed){
                Transfer* transfer = &nodes[node_index].transfer;
                while(transfer != nullptr){
                    int var_index = 0;
                    int next_var = transfer->rhs[var_index];
                    while(next_var != -1){

                        if((new_data & (1 << next_var)) != 0){
                            new_data |= (1 << transfer->x);
                            break;
                        }
                        ++var_index;
                        next_var = transfer->rhs[var_index];
                    }

                    if(transfer->next_transfer_index != -1){
                        transfer = &transfers[transfer->next_transfer_index];
                    }else{
                        break;
                    }
                }


                nodes[node_index].data.data = new_data;
                *has_changed = true;
                is_changed = false;
                // __syncthreads();
            }
        }
    }

}

void bit_cuda::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int extra_transfer_count) {
    Node* dev_nodes = nullptr;
    bool* dev_has_changed = nullptr;
    Transfer* dev_extra_transfers = nullptr;

    int block_count = node_count/128 + 1;
    dim3 threadsPerBlock(128);

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nodes, sizeof(Node)*node_count + 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    if(extra_transfer_count > 0){
        cudaStatus = hipMalloc((void**)&dev_extra_transfers, sizeof(Transfer)*extra_transfer_count);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }
    dev_has_changed = (bool*) (dev_nodes + (sizeof(Node)*node_count));

    cudaStatus = hipMemcpy(dev_nodes, nodes, sizeof(Node)*node_count, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy false to gpu failed");
        goto Error;
    }

    if(extra_transfer_count > 0){
        cudaStatus = hipMemcpy(dev_extra_transfers, transfers, sizeof(Transfer)*extra_transfer_count, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Copy false to gpu failed");
            goto Error;
        }
    }
    // Launch a kernel on the GPU with one thread for each element.
    analyze<<<block_count, threadsPerBlock>>>(dev_nodes, dev_extra_transfers, dev_has_changed, node_count);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(nodes, dev_nodes, sizeof(Node)*node_count, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed with message: %d", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_nodes);

    if(dev_extra_transfers != nullptr){
        hipFree(dev_extra_transfers);
    }

std::cout << "Test1\n";
}

void bit_cuda::execute_analysis_no_transfers(Node* nodes, int node_count){
    execute_analysis(nodes, node_count, nullptr, 0);
}