#include <hipblas.h>
#include <iostream>
#include <assert.h>
#include "GpuManagement.h"
#include <hip/hip_runtime.h>

static hipblasHandle_t handle;

hipblasHandle_t get_cublas(){
    return handle;
}
void create_cublas(){
    hipSetDevice(0);
    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "Create cublas failed" << std::endl;
    }
}
void destroy_cublas(){
    hipblasDestroy(handle);
}

GpuResource::GpuResource(int rowCount, int columnCount, void* data_ptr, size_t element_size){
    this->rowCount = rowCount;
    this->columnCount = columnCount;
    this->element_size = element_size;
    if(hipMalloc(&resource, element_size*rowCount*columnCount) != 0){
        std::cout << "allocation of matrix failed" << std::endl;
    }
    if(hipblasSetMatrix(rowCount, columnCount, element_size, data_ptr, rowCount, resource, rowCount) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "setMatrix of matrix failed" << std::endl;
    }
}

GpuResource::GpuResource(int rowCount, int columnCount, size_t element_size): rowCount(rowCount), columnCount(columnCount){
    this->element_size = element_size;
    if(cublasAlloc(rowCount*columnCount, element_size, &resource) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "allocation of resource failed" << std::endl;
    }
}

GpuResource::GpuResource(const GpuResource& other){
    this->rowCount = other.rowCount;
    this->columnCount = other.columnCount;
    this->element_size = other.element_size;
    size_t size = element_size*rowCount*columnCount;
    if(hipMalloc(&resource, size) != 0){
        std::cout << "allocation of matrix failed" << std::endl;
    }

    if(hipMemcpy(resource, other.resource, size, hipMemcpyDeviceToDevice) != 0){
        std::cout << "Failed to copy gpu matrix to new gpu matrix" << std::endl;
    }
}
GpuResource::GpuResource(GpuResource&& other) noexcept{
    rowCount = other.rowCount;
    other.rowCount = 0;

    columnCount = other.columnCount;
    other.columnCount = 0;

    element_size = other.element_size;
    other.element_size = 0;

    resource = other.resource;
    other.resource = nullptr;
}
GpuResource& GpuResource::operator=(const GpuResource& other){
    //Deallocate old resource first
    hipFree(resource);

    this->rowCount = other.rowCount;
    this->columnCount = other.columnCount;
    this->element_size = other.element_size;
    size_t size = element_size*rowCount*columnCount;
    if(hipMalloc(&resource, size) != 0){
        std::cout << "allocation of matrix failed" << std::endl;
    }

    if(hipMemcpy(resource, other.resource, size, hipMemcpyDeviceToDevice) != 0){
        std::cout << "Failed to copy gpu matrix to new gpu matrix" << std::endl;
    }
    return *this;
}
GpuResource& GpuResource::operator=(GpuResource&& other) noexcept{
    //Deallocate old resource first
    hipFree(resource);

    rowCount = other.rowCount;
    other.rowCount = 0;

    columnCount = other.columnCount;
    other.columnCount = 0;

    element_size = other.element_size;
    other.element_size = 0;

    resource = other.resource;
    other.resource = nullptr;
    return *this;
}

void GpuResource::retrieve_from_gpu(void* dst_ptr){
    auto status = hipblasGetMatrix(rowCount, columnCount, element_size, resource, rowCount, dst_ptr, rowCount);
    if(status != HIPBLAS_STATUS_SUCCESS){
        std::cout << "cublas_get_matrix failed with " << status << std::endl;
    }
}

void GpuResource::multiply_f32_to_f32(const GpuResource& operand, GpuResource& result){
    assert(rowCount == result.rowCount);
    assert(operand.columnCount == result.columnCount);
    assert(columnCount == operand.rowCount);
    float alpha = 1;
    float beta = 0;
    auto status = hipblasSgemm(get_cublas(), HIPBLAS_OP_N, HIPBLAS_OP_N, 
        rowCount, operand.columnCount, columnCount,
        &alpha, (float*)resource, rowCount,
        (float*)operand.resource, operand.rowCount,
        &beta, (float*)result.resource, result.rowCount);

    if(status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Matrix multiplication failed with " << status << std::endl;
    }
}    


void GpuResource::multiply_vector_f32_to_f32(int offset, const GpuResource& operand){
    float* float_resource = (float*)resource;

    float alpha = 1;
    float beta = 0;
    auto status = hipblasSgemv(get_cublas(), HIPBLAS_OP_N, 
        operand.rowCount, operand.columnCount,
        &alpha,
        (float *)operand.resource, operand.rowCount,
        float_resource + offset, 1, 
        &beta, float_resource + offset, 1);

    if(status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Matrix vector multiplication failed with " << status << std::endl;
    }
}

GpuResource::~GpuResource(){
    hipFree(resource);
}


template<>
GpuMatrix<float> GpuMatrix<float>::multiply(GpuMatrix<float>& other) {
    GpuMatrix<float> result(resource.rowCount, resource.columnCount);
    resource.multiply_f32_to_f32(other.resource, result.resource);
    return result;
}

template<>
void GpuMatrix<float>::multiply_vector(int column_index, GpuMatrix<float>& other){
    int offset = column_index * resource.rowCount;
    resource.multiply_vector_f32_to_f32(offset, other.resource);
}