#include <hipblas.h>
#include <iostream>
#include <assert.h>
#include "GpuManagement.h"

static hipblasHandle_t handle;

hipblasHandle_t get_cublas(){
    return handle;
}
void create_cublas(){
    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "Create cublas failed" << std::endl;
    }
}
void destroy_cublas(){
    hipblasDestroy(handle);
}

GpuResource::GpuResource(int rowCount, int columnCount, void* data_ptr, size_t element_size){
    this->rowCount = rowCount;
    this->columnCount = columnCount;
    this->element_size = element_size;
    if(cublasAlloc(rowCount*columnCount, element_size, &resource) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "allocation of matrix failed" << std::endl;
    }

    if(hipblasSetMatrix(rowCount, columnCount, element_size, data_ptr, rowCount, resource, rowCount) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "setMatrix of matrix failed" << std::endl;
    }
}

GpuResource::GpuResource(int rowCount, int columnCount, size_t element_size): rowCount(rowCount), columnCount(columnCount){
    this->element_size = element_size;
    if(cublasAlloc(rowCount*columnCount, element_size, &resource) != HIPBLAS_STATUS_SUCCESS){
        std::cout << "allocation of resource failed" << std::endl;
    }
}

void GpuResource::retrieve_from_gpu(void* dst_ptr){
    std::cout << "Retrieving "<<rowCount << "x"<<columnCount << " with "<<element_size << std::endl;
    auto status = hipblasGetMatrix(rowCount, columnCount, element_size, resource, rowCount, dst_ptr, rowCount);
    if(status != HIPBLAS_STATUS_SUCCESS){
        std::cout << "cublas_get_matrix failed with " << status << std::endl;
    }
}

void GpuResource::multiply_f32_to_f32(const GpuResource& operand, GpuResource& result){
    assert(rowCount == result.rowCount);
    assert(operand.columnCount == result.columnCount);
    assert(columnCount == operand.rowCount);
    float alpha = 1;
    float beta = 0;
    auto status = cublasSgemmEx(get_cublas(), HIPBLAS_OP_N, HIPBLAS_OP_N, 
        operand.rowCount, operand.columnCount, columnCount,
        &alpha, resource, HIP_R_32F, rowCount,
        operand.resource, HIP_R_32F, operand.rowCount,
        &beta, result.resource, HIP_R_32F, result.rowCount);

    if(status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Matrix multiplication failed with " << status << std::endl;
    }
}    


void GpuResource::multiply_vector_f32_to_f32(int offset, const GpuResource& operand){
    
    float* float_resource = (float*)resource;

    float alpha = 1;
    float beta = 0;
    auto status = hipblasSgemv(get_cublas(), HIPBLAS_OP_N, 
        operand.rowCount, operand.columnCount,
        &alpha,
        (float *)operand.resource, operand.rowCount,
        float_resource + offset, 1, 
        &beta, float_resource + offset, 1);

    if(status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Matrix multiplication failed with " << status << std::endl;
    }
}

GpuResource::~GpuResource(){
    cublasFree(resource);
}


template<>
GpuMatrix<float> GpuMatrix<float>::multiply(GpuMatrix<float>& other) {
    GpuMatrix<float> result(resource.rowCount, resource.columnCount);
    resource.multiply_f32_to_f32(other.resource, result.resource);
    return result;
}

template<>
void GpuMatrix<float>::multiply_vector(int column_index, GpuMatrix<float>& other){
    int offset = column_index * resource.columnCount;
    resource.multiply_vector_f32_to_f32(offset, other.resource);
}