#include <hipblas.h>
#include <hip/hip_runtime.h>
void malloc_pinned(void** ptr, unsigned int byte_count){
    hipHostMalloc(ptr, byte_count, hipHostMallocDefault);
}

void free_pinned(void* ptr){
    hipHostFree(ptr);
}