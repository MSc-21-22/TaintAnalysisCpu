#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <vector>
#include <array>

#include "analysis.h"

#define THREAD_COUNT 128

using namespace cuda_worklist;

__device__ void add_sucessors_to_worklist(int* successors, int* work_list, Node* nodes){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    for(int i = 0; i < 5; i++){
        int succ_index = successors[i];
        if (succ_index == -1)
            return;
        intptr_t hash = reinterpret_cast<intptr_t>(&nodes[succ_index]);
        while(work_list[hash % THREAD_COUNT] != -1){
            if(work_list[hash % THREAD_COUNT] == succ_index){
                break;
            }
            hash++;
        }
        work_list[hash % THREAD_COUNT] = succ_index;
    }
}


__global__ void analyze(Node nodes[], int work_columns[][THREAD_COUNT], int work_column_count, Transfer transfers[], int node_count, bool* work_to_do, int i){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    int* work_column = work_columns[i];

    if(node_index < THREAD_COUNT && work_column[node_index] != -1){
        Node& current_node = nodes[work_column[node_index]];
        
        BitVector last = current_node.data;
        BitVector current = current_node.data;

        BitVector joined_data = 0;
        //Join
        {
            int pred_index = 0;
            while (current_node.predecessor_index[pred_index] != -1){
                joined_data |= nodes[current_node.predecessor_index[pred_index]].data;
                ++pred_index;
            }
            current |= joined_data & current_node.join_mask;
        }

        //Transfer
        Transfer* transfer;
        int transfer_index = current_node.first_transfer_index;

        while(transfer_index != -1){
            transfer = &transfers[transfer_index];
            int var_index = 0;
            int next_var = transfer->rhs[var_index];
            while(next_var != -1){

                if((joined_data & (1 << next_var)) != 0){
                    current |= (1 << transfer->x);
                    break;
                }
                ++var_index;
                next_var = transfer->rhs[var_index];
            }
            transfer_index = transfer->next_transfer_index;
        }

        if(last != current){
            current_node.data = current;
            add_sucessors_to_worklist(current_node.successor_index, work_columns[(i+1) % work_column_count], nodes);
            *work_to_do = true;
        }
        work_column[node_index] = -1;   
    }
}

void error(Node* dev_nodes, Transfer* dev_transfers){
    hipFree(dev_nodes);

    if(dev_transfers != nullptr){
        hipFree(dev_transfers);
    }
}

void cuda_worklist::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count) {
    Node* dev_nodes = nullptr;
    bool* dev_work_to_do = nullptr;
    Transfer* dev_transfers = nullptr;
    int** dev_worklists = nullptr;

    for(int i = 0; i < node_count; i++){
        nodes[i].data = 1;
    }
    
    bool work_to_do = true;
    int threadsPerBlock = 128;
    int block_count = THREAD_COUNT/threadsPerBlock;    
    int work_column_count = (node_count + THREAD_COUNT - 1)/THREAD_COUNT;

    std::vector<std::array<int, THREAD_COUNT>> worklists{};
    
    for(int i = 0; i < work_column_count; i++){
        worklists.emplace_back(); 
        for(int j = 0; j < THREAD_COUNT; j++){
            if(i*THREAD_COUNT + j < node_count) {
                worklists[i][j] = i*THREAD_COUNT + j;
            }else{
                worklists[i][j] = -1;
            }
        }
    }

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        error(dev_nodes, dev_transfers);
        return;
    }

    // Allocate work columns
    cudaStatus = hipMalloc((void**)&dev_worklists, sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        error(dev_nodes, dev_transfers);
        return;
    }

    hipMemset(dev_worklists, -1, sizeof(int) * THREAD_COUNT * (work_column_count + 1));

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nodes, sizeof(Node)*node_count + 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        error(dev_nodes, dev_transfers);
        return;
    }


    if(transfer_count > 0){
        cudaStatus = hipMalloc((void**)&dev_transfers, sizeof(Transfer)*transfer_count);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            error(dev_nodes, dev_transfers);
            return;
        }
    }
    dev_work_to_do = (bool*) (dev_nodes + node_count);

    cudaStatus = hipMemcpy(dev_worklists, &worklists[0], sizeof(int) * THREAD_COUNT * work_column_count, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy worklists to gpu failed");
        error(dev_nodes, dev_transfers);
        return;
    }

    cudaStatus = hipMemcpy(dev_nodes, nodes, sizeof(Node)*node_count, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy nodes to gpu failed");
        error(dev_nodes, dev_transfers);
        return;
    }

    if(transfer_count > 0){
        cudaStatus = hipMemcpy(dev_transfers, transfers, sizeof(Transfer)*transfer_count, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Copy transfer functions to gpu failed");
            error(dev_nodes, dev_transfers);
            return;
        }
    }
    
    int i = 0;
    while(work_to_do){
        work_to_do = false;
        cudaStatus = hipMemcpy(dev_work_to_do, &work_to_do, 1, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemset failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Launch a kernel on the GPU with one thread for each element.
        analyze<<<block_count, threadsPerBlock>>>(dev_nodes, (int(*)[THREAD_COUNT])dev_worklists, work_column_count+1, dev_transfers, node_count, dev_work_to_do, i);
        
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }
        
        cudaStatus = hipMemcpy((void*)&work_to_do, dev_work_to_do, sizeof(bool), hipMemcpyDeviceToHost);
        i = (i+1) % (work_column_count+1);
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(nodes, dev_nodes, sizeof(Node)*node_count, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed with message: %d", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_nodes);

    if(dev_transfers != nullptr){
        hipFree(dev_transfers);
    }
}