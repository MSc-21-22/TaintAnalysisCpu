#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <vector>
#include <array>

#include "analysis.h"

using namespace cuda_worklist;

constexpr int THREAD_COUNT = 128*4;

__device__ void add_sucessors_to_worklist(int* successors, int* work_list, Node* nodes){
    for(int i = 0; i < 5; i++){
        int pred_index = successors[i];
        intptr_t hash = (intptr_t)&nodes[pred_index];
        while(work_list[hash % THREAD_COUNT] != -1){
            if(work_list[hash % THREAD_COUNT] == pred_index){
                return;
            }
            hash++;
        }
        work_list[hash] = pred_index;
    }
}


__global__ void analyze(Node nodes[], int work_columns[][THREAD_COUNT], int work_column_count, Transfer transfers[], int node_count, bool* work_to_do){
    int node_index = threadIdx.x + blockDim.x * blockIdx.x;
    int i = 0;
    int* work_column = work_columns[i];

    // check if worklist is empty
    if(node_index == 0)
        *work_to_do = true;

    while(*work_to_do){

        if(node_index == 0)
            *work_to_do = false;

        if(node_index < node_count || work_column[node_index] == -1){
            Node& current_node = nodes[work_column[node_index]];
            current_node.data = 1; // Set taint constant to true
            
            bool is_changed = true;
            BitVector last_joined = 0;
            BitVector current = current_node.data;

            BitVector joined_data = 0;
            //Join
            {
                int pred_index = 0;
                while (current_node.predecessor_index[pred_index] != -1){
                    joined_data |= nodes[current_node.predecessor_index[pred_index]].data;
                    ++pred_index;
                }

                is_changed |= last_joined != joined_data;
                last_joined = joined_data;
                current |= joined_data & current_node.join_mask;
            }

            //Transfer
            if(is_changed){
                Transfer* transfer;
                int transfer_index = current_node.first_transfer_index;

                while(transfer_index != -1){
                    transfer = &transfers[transfer_index];
                    int var_index = 0;
                    int next_var = transfer->rhs[var_index];
                    while(next_var != -1){

                        if((joined_data & (1 << next_var)) != 0){
                            current |= (1 << transfer->x);
                            break;
                        }
                        ++var_index;
                        next_var = transfer->rhs[var_index];
                    }
                    transfer_index = transfer->next_transfer_index;
                }

                current_node.data = current;

                add_sucessors_to_worklist(current_node.successor_index, work_columns[(i+1) % work_column_count], nodes);
                *work_to_do = true;
            }

        }
        __syncthreads();
        __threadfence();
        i = (i+1) % work_column_count;
    }
}

void cuda_worklist::execute_analysis(Node* nodes, int node_count, Transfer* transfers, int transfer_count) {
    Node* dev_nodes = nullptr;
    bool* work_to_do = nullptr;
    Transfer* dev_transfers = nullptr;
    int** dev_worklists = nullptr;

    
    int threadsPerBlock = 128;
    int block_count = THREAD_COUNT/threadsPerBlock;    
    int work_column_count = node_count/THREAD_COUNT;

    std::vector<std::array<int, THREAD_COUNT>> worklists{};
    
    for(int i = 0; i < work_column_count; i++){
        worklists.emplace_back(); 
        for(int j = 0; j < THREAD_COUNT; j++){
            worklists[i][j] = i*THREAD_COUNT + j;
        }
    }

    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate work columns
    cudaStatus = hipMalloc((void**)&dev_worklists, sizeof(int) * THREAD_COUNT * (work_column_count + 1));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemset(dev_worklists, -1, sizeof(int) * THREAD_COUNT * (work_column_count + 1));

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nodes, sizeof(Node)*node_count + 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    if(transfer_count > 0){
        cudaStatus = hipMalloc((void**)&dev_transfers, sizeof(Transfer)*transfer_count);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }
    work_to_do = (bool*) (dev_nodes + (sizeof(Node)*node_count));

    cudaStatus = hipMemcpy(dev_worklists, &worklists[0], sizeof(int) * THREAD_COUNT * work_column_count, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy worklists to gpu failed");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_nodes, nodes, sizeof(Node)*node_count, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy nodes to gpu failed");
        goto Error;
    }

    if(transfer_count > 0){
        cudaStatus = hipMemcpy(dev_transfers, transfers, sizeof(Transfer)*transfer_count, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Copy transfer functions to gpu failed");
            goto Error;
        }
    }
    // Launch a kernel on the GPU with one thread for each element.
    analyze<<<block_count, threadsPerBlock>>>(dev_nodes, (int(*)[THREAD_COUNT])dev_worklists, work_column_count, dev_transfers, node_count, work_to_do);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(nodes, dev_nodes, sizeof(Node)*node_count, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed with message: %d", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_nodes);

    if(dev_transfers != nullptr){
        hipFree(dev_transfers);
    }
}