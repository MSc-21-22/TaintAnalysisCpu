#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <iostream>

#include <stdio.h>

bool cudaMemcmp(const float *a, const float* b, int size);

__global__ void memcmp_kernel(const float* a, const float* b, int size, bool* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size && ((a[i] !=0) != (b[i] != 0))){
        *result = true;
    }
}

bool gpu_mem_cmp(GpuResource& a, GpuResource& b){
    int size = a.rowCount*a.columnCount;
    return cudaMemcmp((float*)a.resource, (float*)b.resource, size);
}

// Helper function for using CUDA to add vectors in parallel.
bool cudaMemcmp(const float *a, const float* b, int size)
{
    bool* dev_result = nullptr;
    bool result = false;
    int block_count = size/128 + 1;
    dim3 threadsPerBlock(128);

    // Choose which GPU to run on, change this on a multi-GPU system.
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_result, sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_result, &result, sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy false to gpu failed");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    memcmp_kernel<<<block_count, threadsPerBlock>>>(a, b, size, dev_result);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed with message: %d", cudaStatus);
        goto Error;
    }

    hipFree(dev_result);
    return !result;

Error:
    hipFree(dev_result);
    
    return false;
}
